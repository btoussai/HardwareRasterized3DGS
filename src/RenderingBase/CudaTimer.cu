#include "hip/hip_runtime.h"
//
// Created by Briac on 03/07/2025.
//

#include "CudaTimer.cuh"

#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

CudaTimer::CudaTimer() {
    checkCudaErrors(hipEventCreate(&start_time));
    checkCudaErrors(hipEventCreate(&stop_time));
}

CudaTimer::~CudaTimer() {
    checkCudaErrors(hipEventDestroy(start_time));
    checkCudaErrors(hipEventDestroy(stop_time));
}

void CudaTimer::start() {
    checkCudaErrors(hipEventRecord(start_time));
}

void CudaTimer::stop() {
    checkCudaErrors(hipEventRecord(stop_time));
    new_measure = true;
}

float CudaTimer::getTimeMs() {
    float elapsedTime_ms = 0.0f;
    auto error = hipEventElapsedTime(&elapsedTime_ms, start_time, stop_time);
    if(error == hipErrorInvalidHandle){
        // timer not started yet.
    }else if(error == hipErrorNotReady){
        // result not yet available
    }else{
        checkCudaErrors(error);
    }

    // ema
    const double beta = 0.99;
    if(elapsedTime_ms > 0.0f && new_measure){
        calls++;
        total_ms = total_ms * beta + elapsedTime_ms * (1.0 - beta);
        new_measure = false;
    }
    const double normalization = 1.0 / (1.0 - pow(beta, calls));
    return calls > 0 ? float(total_ms * normalization) : 0.0f;
}
