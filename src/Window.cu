//
// Created by Briac on 18/06/2025.
//

#include "Window.cuh"

#include <iostream>
#include <cstdint>

#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include "./RenderingBase/hip/hip_runtime_api.h"
#include "imgui/imgui.h"
#include "imgui/imgui_impl_glfw.h"
#include "imgui/imgui_impl_opengl3.h"

#include "RenderingBase/Camera.h"
#include "RenderingBase/GLShaderLoader.h"
#include "RenderingBase/GLIntrospection.h"
#include "RenderingBase/CudaIntrospection.cuh"

#include "PointCloudLoader.h"

#include <thread>
#include <chrono>

static void error_callback(int error, const char *description) {
    fprintf(stderr, "Error: %s\n", description);
    fflush(stderr);
}

static void myGlDebugCallback(GLenum source,
                              GLenum type,
                              GLuint id,
                              GLenum severity,
                              GLsizei length,
                              const GLchar *message,
                              const void *userParam){

    if(severity == GL_DEBUG_SEVERITY_HIGH || severity == GL_DEBUG_SEVERITY_MEDIUM
    || severity == GL_DEBUG_SEVERITY_LOW){
        std::cout <<"GL_DEBUG: " <<message <<std::endl;
    }


}
static void glad_callback_custom(void *ret, const char *name, GLADapiproc apiproc, int len_args, ...) {
    GLenum error_code;

    error_code = glad_glGetError();

    if (error_code != GL_NO_ERROR) {
        std::string type("UNKNOWN");
        if (error_code == GL_INVALID_ENUM) {
            type = "GL_INVALID_ENUM";
        } else if (error_code == GL_INVALID_OPERATION) {
            type = "GL_INVALID_OPERATION";
        } else if (error_code == GL_INVALID_VALUE) {
            type = "GL_INVALID_VALUE";
        } else if (error_code == GL_INVALID_INDEX) {
            type = "GL_INVALID_INDEX";
        } else if (error_code == GL_INVALID_FRAMEBUFFER_OPERATION) {
            type = "GL_INVALID_FRAMEBUFFER_OPERATION";
        } else if (error_code == GL_OUT_OF_MEMORY) {
            type = "GL_OUT_OF_MEMORY";
        } else if(error_code == GL_CONTEXT_LOST){
            type = "GL_CONTEXT_LOST";
        }

        std::cout << "ERROR " << error_code << " in " << name << " (" << type
                  << ")" << std::endl;

        if (error_code == GL_OUT_OF_MEMORY) {
            throw std::string("OpenGL Fatal Error: Out of memory");
        }
    }
}

static void framebuffer_size_callback(GLFWwindow *window, int width,
                                      int height) {
    glViewport(0, 0, width, height);
}

static double scroll;
static void scroll_callback(GLFWwindow *window, double xoffset,
                            double yoffset) {
    scroll = yoffset;
}

Window::Window(const std::string &title, int samples) {
    if (!glfwInit())
        throw "Error while initializing GLFW";

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GLFW_FALSE);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_API, GLFW_TRUE);
    glfwWindowHint(GLFW_SAMPLES, samples);
    glfwWindowHint(GLFW_MAXIMIZED, GLFW_TRUE);
    glfwWindowHint(GLFW_FOCUS_ON_SHOW, GLFW_TRUE);
//    glfwWindowHint(GLFW_CONTEXT_RELEASE_BEHAVIOR, GLFW_RELEASE_BEHAVIOR_FLUSH);
    glfwWindowHint(GLFW_CONTEXT_RELEASE_BEHAVIOR, GLFW_RELEASE_BEHAVIOR_NONE);

    glfwWindowHint(GLFW_CONTEXT_ROBUSTNESS, GLFW_LOSE_CONTEXT_ON_RESET);
    glfwWindowHint(GLFW_OPENGL_DEBUG_CONTEXT, GLFW_TRUE);
//    glfwWindowHint(GLFW_CONTEXT_NO_ERROR, GLFW_FALSE);

//    glfwWindowHint(GLFW_VISIBLE, GLFW_FALSE);
//    glfwWindowHint(GLFW_FOCUS_ON_SHOW, GLFW_FALSE);

    w = glfwCreateWindow(800, 600, title.c_str(), NULL, NULL);
    if (!w) {
        glfwTerminate();
        throw std::string("Error while creating the window");
    }

    glfwMakeContextCurrent(w);
    gladLoadGL(glfwGetProcAddress);
    gladInstallGLDebug();

    std::cout << "Version: " << glGetString(GL_VERSION) << std::endl;
    std::cout << "Vendor: " << glGetString(GL_VENDOR) << std::endl;
    std::cout << "Renderer: " << glGetString(GL_RENDERER) << std::endl;
    std::cout << "GLSL Version: " << glGetString(GL_SHADING_LANGUAGE_VERSION)
              << std::endl;

    glfwSetErrorCallback(error_callback);
    glfwSetFramebufferSizeCallback(w, framebuffer_size_callback);
    glfwSetScrollCallback(w, scroll_callback);

    glEnable(GL_MULTISAMPLE);
    glEnable(GL_TEXTURE_CUBE_MAP_SEAMLESS);

    gladSetGLPostCallback(reinterpret_cast<GLADpostcallback>(glad_callback_custom));
    glDebugMessageCallback(myGlDebugCallback, nullptr);

//    guis = std::make_unique<GUIs>();
//    guis->init_IMGUI(w);
//    reloadFonts();
//
//    setTitle();
//    toogleFullscreen();
//    toogleVsync();
    auto ctx = ImGui::CreateContext();
    ImGui::SetCurrentContext(ctx);
    ImGui_ImplOpenGL3_Init();
    ImGui_ImplGlfw_InitForOpenGL(w, true);
}

Window::~Window() {

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    glfwTerminate();
}

void loadHeaders(){

    std::function<void(std::unordered_map<std::string, std::string>&)> replacements;
    std::unordered_map<std::string, std::string> m;

    // glsl syntax hacks
    m["___flat"] = "flat";
    m["___out"] = "out";
    m["___in"] = "in";
    m["___inout"] = "inout";
    m["___discard"] = "discard";
    m["//--"] = "";
    m["/\\*--"] = "";
    m["--\\*/"] = "";

    m["#include"] = "";
    m["\\bstatic\\b"] = "";
    m["\\binline\\b"] = "";
    m["__UNKOWN_SIZE"] = "";


    std::string regex_str = "";
    for(const auto& [s, d] : m){
        if(regex_str.size() > 0){
            regex_str += "|" + s;
        }else{
            regex_str += s;
        }
    }
    std::regex re = std::regex(regex_str);

    std::vector<std::string> headers;
    headers.push_back("resources/shaders/common/GLSLDefines.h");
    headers.push_back("resources/shaders/common/CommonTypes.h");
    headers.push_back("resources/shaders/common/Uniforms.h");
    headers.push_back("resources/shaders/common/Covariance.h");
    GLShaderLoader::instance->loadHeaders(headers, m, re);
}

void Window::mainloop(int argc, char **argv) {

    Camera camera;
    GLShaderLoader shaderLoader("resources/shaders", "SparseVoxelReconstruction");
    loadHeaders();

    unsigned int gl_device_count;
    int gl_device_id;
    checkCudaErrors(hipGLGetDevices(&gl_device_count, &gl_device_id, 1, hipGLDeviceListAll));
    int cuda_device_id = gl_device_id;
    checkCudaErrors(hipSetDevice(cuda_device_id));

    hipDeviceProp_t props;
    checkCudaErrors(hipGetDeviceProperties(&props, gl_device_id));
    printf("GL   : %-24s (%2d SMs)\n", props.name, props.multiProcessorCount);
    checkCudaErrors(hipGetDeviceProperties(&props, cuda_device_id));
    printf("CUDA : %-24s (%2d SMs)\n", props.name, props.multiProcessorCount);

//    CudaBufferSetupBoundsCheck();

    GaussianCloud cloud;
    cloud.initShaders();

    bool windowHovered = false;
    while (!glfwWindowShouldClose(this->w)) {
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::BeginMainMenuBar();
        GLIntrospection::inspectObjects();
        CudaIntrospection::inspectBuffers();
        ImGui::EndMainMenuBar();

        ImGui::Begin("Window");
//        ImGui::ShowMetricsWindow();

        if(ImGui::Button("Reload Shaders")){
            shaderLoader.checkForFileUpdates();
        }
        if(ImGui::Button("Load ply")){
            PointCloudLoader::load(cloud, "bicycle.ply", true);
        }
        if(cloud.initialized){
            ImGui::Text("The point cloud contains %d 3D gaussians.", cloud.num_gaussians);
        }

        camera.updateView(w, windowHovered, (float)scroll);


        int width, height;
        glfwGetWindowSize(w, &width, &height);
        glfwGetFramebufferSize(w, &width, &height);

        glViewport(0, 0, width, height); // reset the viewport
//        glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
        // need to clear with alpha = 1 for front to back blending
        glClearColor(0.0f,0.0f,0.0f,1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
//        windowHovered = ImGui::IsWindowHovered(ImGuiHoveredFlags_AnyWindow);

        if(cloud.initialized){
            cloud.GUI(camera);
            cloud.render(camera);
        }

        windowHovered = ImGui::GetIO().WantCaptureMouse;

        ImGui::End();

        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        scroll *= 0.90f;

        glfwSwapBuffers(this->w);
        glfwPollEvents();

        std::this_thread::sleep_for(std::chrono::milliseconds(10));

        if(windowHovered){
            scroll = 0;
        }

    }
}